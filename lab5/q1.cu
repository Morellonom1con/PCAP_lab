#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__global__ void vectorAdd(float *A, float *B, float *C, int N) {
    int idx = blockIdx.x * blockDim.x + t hreadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    int N;
    
    printf("Enter the length of the vectors: \n");
    scanf("%d",&N);
    int size=N*sizeof(float);

    float h_A[N],h_B[N],h_C[N];

    printf("Enter vector A: \n");
    for (int i = 0; i < N; i++) {
        scanf("%f",&h_A[i]);
    }

    printf("Enter vector B: \n");
    for (int i = 0; i < N; i++) {
        scanf("%f",&h_B[i]);
    }

    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int blockSize = N;// for b) its some blocksize say 256
    int gridSize = 1;// and gridsize (N-1)/blocksize+1

    vectorAdd<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("\nVector C :\n");
    for (int i=0; i< N;i++)
    {
        printf("%f ",h_C[i]);
    }
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
