#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<math.h>

__global__ void Sine(float *A, float *C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        C[idx] = sin(A[idx]);
    }
}

int main() {
    int N;
    
    printf("Enter the number of angles: \n");
    scanf("%d",&N);
    int size=N*sizeof(float);

    float h_A[N],h_C[N];

    printf("Enter Array of radian angles: \n");
    for (int i = 0; i < N; i++) {
        scanf("%f",&h_A[i]);
    }

    float *d_A, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (N-1)/blockSize+1;

    Sine<<<gridSize, blockSize>>>(d_A, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("\nSine of angles :\n");
    for (int i=0; i< N;i++)
    {
        printf("%f ",h_C[i]);
    }
    hipFree(d_A);
    hipFree(d_C);
    return 0;
}
