#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void newstring(const char *inp, char *ans, int l) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < l) {
        int copy = l - i;
        int j = (l * (l + 1)) / 2 - (copy * (copy + 1)) / 2;
        for (int k = 0; k < copy; ++k)
            ans[j + k] = inp[k];
    }
}

int main() {
    int l;
    printf("Enter word length: ");
    scanf("%d", &l);
    char *arr = (char *)malloc(l + 1);
    printf("Enter word: ");
    scanf("%s", arr);
    int total = (l * (l + 1)) / 2;
    char *ans = (char *)malloc(total + 1);
    char *darr, *dans;
    hipMalloc(&darr, l + 1);
    hipMalloc(&dans, total + 1);
    hipMemcpy(darr, arr, l + 1, hipMemcpyHostToDevice);
    newstring<<<(l + 255) / 256, 256>>>(darr, dans, l);
    hipMemcpy(ans, dans, total + 1, hipMemcpyDeviceToHost);
    ans[total] = '\0';
    printf("%s\n", ans);
    free(arr); free(ans);
    hipFree(darr); hipFree(dans);
    return 0;
}
