#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void find(const char *arr, int la, const char *word, int lw, int *ans) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i + lw <= la) {
        bool match = true;
        for (int j = 0; j < lw; ++j) {
            if (arr[i + j] != word[j]) {
                match = false;
                break;
            }
        }
        if (match) atomicAdd(ans, 1);
    }
}

int main() {
    int la, lw, ans = 0;
    char *arr, *word, *darr, *dword;
    int *dans;

    printf("Enter string length: ");
    scanf("%d%*c", &la);
    arr = (char *)malloc(la + 1);
    printf("Enter string: ");
    fgets(arr, la + 1, stdin);

    printf("Enter word length: ");
    scanf("%d%*c", &lw);
    word = (char *)malloc(lw + 1);
    printf("Enter word: ");
    fgets(word, lw + 1, stdin);

    hipMalloc(&darr, la + 1);
    hipMalloc(&dword, lw + 1);
    hipMalloc(&dans, sizeof(int));
    hipMemcpy(darr, arr, la + 1, hipMemcpyHostToDevice);
    hipMemcpy(dword, word, lw + 1, hipMemcpyHostToDevice);
    hipMemset(dans, 0, sizeof(int));

    int threads = 256, blocks = (la + threads - 1) / threads;
    find<<<blocks, threads>>>(darr, la, dword, lw, dans);

    hipMemcpy(&ans, dans, sizeof(int), hipMemcpyDeviceToHost);

    printf("Occurrences: %d\n", ans);

    free(arr); free(word);
    hipFree(darr); hipFree(dword); hipFree(dans);
    return 0;
}
